#include "hip/hip_runtime.h"
// This program executes a typical Interval Join
#include <iostream>
#include <omp.h>
#include <time.h>
#include "intervalJoin.h"
using namespace std;


struct node 
{ 
	int middle;
    int *start,*end,*index; 
    int length;
    struct node *left, *right; 
}; 
   
struct node *newNode(int middle,int *start, int *end, int start_idx,int end_idx) 
{ 
	int i;
    struct node *temp =  (struct node *)malloc(sizeof(struct node)); 
    temp->middle=middle;
    temp->length=end_idx-start_idx+1;
    temp->start = (int*)malloc(temp->length*sizeof(int));
	temp->end = (int*)malloc(temp->length*sizeof(int));
	temp->index = (int*)malloc(temp->length*sizeof(int));
	for(i=0;i<temp->length;i++){
		temp->start[i]=start[start_idx+i];
		temp->end[i]=end[start_idx+i];
		temp->index[i]=start_idx+i;
	} 
	temp->left = temp->right = NULL; 
    return temp; 
}
   
void search(struct node *node, int start, int end, int index) 
{ 
	int i;
    if (node != NULL) 
    { 
    	for(i=0;i<node->length;i++){
    		if((node->start[i]<=start && start<=node->end[i]) || (node->start[i]<=end && end<=node->end[i]) || (node->start[i]<=start && end<=node->end[i]) || (node->start[i]>=start && end>=node->end[i])){
    			if(node->index[i]<start_index[index])
    				start_index[index]=node->index[i];
    			if(node->index[i]>end_index[index])
    				end_index[index]=node->index[i];  				
			}				
		}
    	if(start<=node->middle && node->middle<=end){
        	search(node->left,start,end, index); 
        	search(node->right,start,end, index);	
		}
        else if(end<node->middle)
        	search(node->left,start,end, index);
        else if(node->middle<start)
        	search(node->right,start,end, index);
    }
} 


struct node* make_tree(struct node* node,int *input_start, int *input_end,int array_start, int array_end){
	int i;
	int middle;
	int start_idx=-1,end_idx=-1;
	if(array_start<=array_end){
		middle=(input_start[array_start]+input_end[array_end])/2;
		for(i=array_start;i<=array_end;i++){
			if(input_start[i]<=middle && middle<=input_end[i]){
				if(start_idx==-1){
					start_idx=i;
				}
				end_idx=i;
			}
		}
		i=0;
		if(start_idx==-1 && end_idx==-1){
			while(input_end[i]<=middle)
				i++;
			start_idx=i-1;
			while(input_start[i]<middle)
				i++;
			end_idx=i-1;
		}
   
		node=newNode(middle, input_start,input_end,start_idx,end_idx);
	}
	if(start_idx>=0 && array_start>=0 && start_idx-1>=array_start)
		node->left=make_tree(node,input_start, input_end,array_start,start_idx-1);
	
			
	if(start_idx>=0 && end_idx+1<=array_end)
		node->right=make_tree(node,input_start, input_end,end_idx+1,array_end);
	
	return node;	
}


// This is the CPU version, please don't modify it
void intervalJoinCPU(int id)
{
	int i;
	struct node* root=NULL;
	int search_size= setB.length[id] * sizeof(int);
    start_index=(int*)malloc(search_size);
    end_index=(int*)malloc(search_size);
	
    root=make_tree(root,inStartA,inEndA,0,setA.length[id]-1);
    //inorder(root);
	#pragma omp parallel for
	for(i=0;i<setB.length[id];i++){
        start_index[i]=INT_MAX;
        end_index[i]=INT_MIN;
		search(root,inStartB[i],inEndB[i],i);
        outCPU_Begin[i]=start_index[i];
        outCPU_End[i]=end_index[i];
		//cout<<i<<endl;
    }
	int total_intersects=0;
        for(i=0;i<setB.length[id];i++){
                if(outCPU_Begin[i]!=INT_MAX && outCPU_End[i]!=INT_MIN){
                        total_intersects+=(outCPU_End[i]-outCPU_Begin[i]+1);
        }
    }
	//cout<<total_intersects<<endl;
	
	free(start_index);
	free(end_index);
}

// This is the CPU version, please don't modify it
void executeQuery_CPU(int id, int min_overlap)
{
	int index_first,index_last;
	int total_count=0;
	
	for(int i=0;i<setB.length[id];i++){
		if(outCPU_End[i]-outCPU_Begin[i]+1>=min_overlap){
			for(int k=outCPU_Begin[i];k<=outCPU_End[i];k++){
				index_first=abs(inStartA[k]-inStartB[i])%4;
				index_last=abs(inEndA[k]-inEndB[i])%4;
				
				if(inStringA[k][index_first]==inStringB[i][index_last]){
					outCPU_count[i]++;
				}
			}
			total_count+=outCPU_count[i];
			
		}
	}
}

/***	Implement your CUDA Kernel here	***/
__global__
void intervalJoinGPU()
{
}
/***	Implement your CUDA Kernel here	***/

/***    Implement your CUDA Kernel here ***/
__global__
void sort()
{
}
/***    Implement your CUDA Kernel here ***/

/***    Implement your CUDA Kernel here ***/
__global__
void executeQuery_GPU(int id)
{
}
/***    Implement your CUDA Kernel here ***/


int main()
{
	int i;
	timespec time_begin, time_end;
	int intervalJoinCPUExecTime, intervalJoinGPUExecTime;
	int cpuTotalTime=0,gpuTotalTime=0; 
	FILE *fpA, *fpB;
	read_Meta();
	
	fpA = fopen ("data/dataA.csv","r");
	fpB = fopen ("data/dataB.csv","r");
	
	for(i=0;i<setA.count;i++){
		init_from_csv(fpA, fpB, i);
		
		intervalJoinCPU(i);
		
		clock_gettime(CLOCK_REALTIME, &time_begin);
		executeQuery_CPU(i,2);
		
		clock_gettime(CLOCK_REALTIME, &time_end);
		intervalJoinCPUExecTime = timespec_diff_us(time_begin, time_end);
		cout << "CPU time for executing a typical Query = " <<  intervalJoinCPUExecTime / 1000 << "ms" << endl;
		cpuTotalTime+=intervalJoinCPUExecTime;
		
		randomize (inStartA,inEndA,inStringA, setA.length[i],4);
		
		
		//Do the required GPU Memory allocation here
		
		//Do the required GPU Memory allocation here
		
		//Configure the CUDA Kernel call here
		sort<<<1,1>>>(); // Lunch the kernel
		
		clock_gettime(CLOCK_REALTIME, &time_begin);
		executeQuery_GPU<<<1,1>>>(i);  // Lunch the kernel
		hipDeviceSynchronize(); // Do synchronization before clock_gettime()
		//Copy back the result from GPU Memory to CPU memory array outGPU_count
		
		//Copy back the result from GPU Memory to CPU memory array outGPU_count
		
		clock_gettime(CLOCK_REALTIME, &time_end);
		intervalJoinGPUExecTime = timespec_diff_us(time_begin, time_end);
		cout << "GPU time for executing a typical Query = " << intervalJoinGPUExecTime / 1000 << "ms" << endl;
		cpuTotalTime+=intervalJoinGPUExecTime;
		
		if(checker(setB.length[i])){
			cout << "Congratulations! You pass the check." << endl;
			cout << "Speedup: " << (float)intervalJoinCPUExecTime / intervalJoinGPUExecTime << endl;
		}
		else
			cout << "Sorry! Your result is wrong." << endl;
		
		ending(i);
	}
	
	fclose(fpA);
	fclose(fpB);

	return 0;
}
